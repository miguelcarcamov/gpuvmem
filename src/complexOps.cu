#include "hip/hip_runtime.h"
#include "complexOps.cuh"

__host__ __device__ hipfftComplex floatComplexZero() {
  hipfftComplex zero = make_hipFloatComplex(0.0f, 0.0f);
  return zero;
};

__host__ __device__ hipfftDoubleComplex doubleComplexZero() {
  hipfftDoubleComplex zero = make_hipDoubleComplex(0.0, 0.0);
  return zero;
};

__host__ __device__ float amplitude(hipfftComplex c) {
  float amp = hipCabsf(c);
  return amp;
};

__host__ __device__ double amplitude(hipfftDoubleComplex c) {
  double amp = hipCabs(c);
  return amp;
};

__host__ __device__ float phaseDegrees(hipfftComplex c) {
  float phase = atan2f(c.y, c.x) * 180.0f / HIP_PI_F;
  return phase;
};

__host__ __device__ double phaseDegrees(hipfftDoubleComplex c) {
  double phase = atan2(c.y, c.x) * 180.0 / HIP_PI;
  return phase;
};

__host__ __device__ hipfftComplex mulComplexReal(hipfftComplex c1, float c2) {
  hipfftComplex result;
  result = hipCmulf(c1, make_hipFloatComplex(c2, 0.0f));
  result.x = c1.x * c2;
  result.y = c1.y * c2;

  return result;
};

__host__ __device__ hipfftDoubleComplex mulComplexReal(hipfftDoubleComplex c1,
                                                      float c2) {
  hipfftDoubleComplex result;
  result = hipCmul(c1, make_hipDoubleComplex(c2, 0.0));
  return result;
};

__host__ __device__ hipfftComplex divComplexReal(hipfftComplex c1, float c2) {
  hipfftComplex result;

  result = hipCdivf(c1, make_hipFloatComplex(c2, 0.0));
  return result;
};

__host__ __device__ hipfftDoubleComplex divComplexReal(hipfftDoubleComplex c1,
                                                      double c2) {
  hipfftDoubleComplex result;
  result = hipCdiv(c1, make_hipDoubleComplex(c2, 0.0));
  return result;
};

__global__ void mulArrayComplexComplex(hipfftComplex *c1, hipfftComplex *c2,
                                       int M, int N) {
  const int i = threadIdx.y + blockDim.y * blockIdx.y;
  const int j = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < M && j < N) {
    c1[N * i + j] = hipCmulf(c1[N * i + j], c2[N * i + j]);
  }
};

__global__ void mulArrayComplexComplex(hipfftDoubleComplex *c1,
                                       hipfftDoubleComplex *c2, int M, int N) {
  const int i = threadIdx.y + blockDim.y * blockIdx.y;
  const int j = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < M && j < N) {
    c1[N * i + j] = hipCmul(c1[N * i + j], c2[N * i + j]);
  }
};
