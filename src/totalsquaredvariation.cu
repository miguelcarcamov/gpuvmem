#include "totalsquaredvariation.cuh"

extern long M, N;
extern int image_count;
extern float* penalizators;
extern int nPenalizators;

TotalSquaredVariationP::TotalSquaredVariationP() {
  this->name = "Total Squared Variation";
};

float TotalSquaredVariationP::calcFi(float* p) {
  float result = 0.0f;
  this->set_fivalue(TotalSquaredVariation(p, device_S, penalization_factor, mod,
                                          order, imageIndex, this->iteration));
  result = (penalization_factor) * (this->get_fivalue());
  return result;
}
void TotalSquaredVariationP::calcGi(float* p, float* xi) {
  DTSVariation(p, device_DS, penalization_factor, mod, order, imageIndex,
               this->iteration);
};

void TotalSquaredVariationP::restartDGi() {
  checkCudaErrors(hipMemset(device_DS, 0, sizeof(float) * M * N));
};

void TotalSquaredVariationP::addToDphi(float* device_dphi) {
  linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void TotalSquaredVariationP::setSandDs(float* S, float* Ds) {
  hipFree(this->device_S);
  hipFree(this->device_DS);
  this->device_S = S;
  this->device_DS = Ds;
};

namespace {
Fi* CreateTotalSquaredVariation() {
  return new TotalSquaredVariationP;
}

const std::string name = "TotalSquaredVariation";
const bool RegisteredTotalSquaredVariation =
    registerCreationFunction<Fi, std::string>(name,
                                              CreateTotalSquaredVariation);
};  // namespace
