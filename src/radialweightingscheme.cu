#include "hip/hip_runtime.h"
#include "radialweightingscheme.cuh"

RadialWeightingScheme::RadialWeightingScheme() : WeightingScheme(){};

RadialWeightingScheme::RadialWeightingScheme(int threads)
    : WeightingScheme(threads){};

void RadialWeightingScheme::apply(std::vector<MSDataset>& d) {
  std::cout << "Running weighting scheme with " << this->threads << " threads"
            << std::endl;
  float w;
  double3 uvw;
  for (int j = 0; j < d.size(); j++) {
    for (int f = 0; f < d[j].data.nfields; f++) {
      for (int i = 0; i < d[j].data.total_frequencies; i++) {
        for (int s = 0; s < d[j].data.nstokes; s++) {
          d[j].fields[f].backup_visibilities[i][s].weight.resize(
              d[j].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);

#pragma omp parallel for schedule(static, 1) \
    num_threads(this->threads) private(uvw, w)
          for (int z = 0;
               z < d[j].fields[f].numVisibilitiesPerFreqPerStoke[i][s]; z++) {
            uvw = d[j].fields[f].visibilities[i][s].uvw[z];
            uvw.x = metres_to_lambda(uvw.x, d[j].fields[f].nu[i]);
            uvw.y = metres_to_lambda(uvw.y, d[j].fields[f].nu[i]);
            w = d[j].fields[f].visibilities[i][s].weight[z];
            d[j].fields[f].visibilities[i][s].weight[z] *=
                distance(uvw.x, uvw.y, 0.0f, 0.0f);

            if (NULL != this->uvtaper)
              d[j].fields[f].visibilities[i][s].weight[z] *=
                  this->uvtaper->getValue(uvw.x, uvw.y);

            if (this->modify_weights)
              d[j].fields[f].backup_visibilities[i][s].weight[z] =
                  d[j].fields[f].visibilities[i][s].weight[z];
            else
              d[j].fields[f].backup_visibilities[i][s].weight[z] = w;
          }
        }
      }
    }
  }
};

namespace {
WeightingScheme* CreateWeightingScheme() {
  return new RadialWeightingScheme;
}

const std::string name = "Radial";
const bool RegisteredRadialWeighting =
    registerCreationFunction<WeightingScheme, std::string>(
        name,
        CreateWeightingScheme);
};  // namespace
