#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
   Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
   Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

   This program includes Numerical Recipes (NR) based routines whose
   copyright is held by the NR authors. If NR routines are included,
   you are required to comply with the licensing set forth there.

   Part of the program also relies on an an ANSI C library for multi-stream
   random number generation from the related Prentice-Hall textbook
   Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
   for more information please contact leemis@math.wm.edu

   Additionally, this program uses some NVIDIA routines whose copyright is held
   by NVIDIA end user license agreement (EULA).

   For the original parts of this code, the following license applies:

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program. If not, see <http://www.gnu.org/licenses/>.
 * -------------------------------------------------------------------------
 */

#include "brent.cuh"
#include "nrutil.h"
#define ITMAX 500
#define CGOLD 0.3819660
#define ZEPS 1.0e-10
#define SHFT(a, b, c, d) \
  (a) = (b);             \
  (b) = (c);             \
  (c) = (d);
__host__ float brent(float ax, float bx, float cx, float tol, float *xmin,
                     float (*f)(float)) {
  float a, b, d, etemp, fu, fv, fw, fx, p, q, r, tol1, tol2, u, v, w, x, xm;
  float e = 0.0;

  a = (ax < cx ? ax : cx);
  b = (ax > cx ? ax : cx);
  x = w = v = bx;
  fw = fv = fx = (*f)(x);
  for (int iter = 1; iter <= ITMAX; iter++) {
    xm = 0.5 * (a + b);
    tol2 = 2.0 * (tol1 = tol * fabs(x) + ZEPS);
    if (fabs(x - xm) <= (tol2 - 0.5 * (b - a))) {
      *xmin = x;
      return fx;
    }
    if (fabs(e) > tol1) {
      r = (x - w) * (fx - fv);
      q = (x - v) * (fx - fw);
      p = (x - v) * q - (x - w) * r;
      q = 2.0 * (q - r);

      if (q > 0.0) {
        p = -p;
      }

      q = fabs(q);
      etemp = e;
      e = d;

      if (fabs(p) >= fabs(0.5 * q * etemp) || p <= q * (a - x) ||
          p >= q * (b - x))
        d = CGOLD * (e = (x >= xm ? a - x : b - x));
      else {
        d = p / q;
        u = x + d;
        if (u - a < tol2 || b - u < tol2) {
          d = SIGN(tol1, xm - x);
        }
      }
    } else {
      d = CGOLD * (e = (x >= xm ? a - x : b - x));
    }
    u = (fabs(d) >= tol1 ? x + d : x + SIGN(tol1, d));
    fu = (*f)(u);
    if (fu <= fx) {
      if (u >= x) {
        a = x;
      } else {
        b = x;
      }
      SHFT(v, w, x, u)
      SHFT(fv, fw, fx, fu)
    } else {
      if (u < x) {
        a = u;
      } else {
        b = u;
      }

      if (fu <= fw || w == x) {
        v = w;
        w = u;
        fv = fw;
        fw = fu;
      } else if (fu <= fv || v == x || v == w) {
        v = u;
        fv = fu;
      }
    }
  }
  printf("Too many iterations in brent\n");
  *xmin = x;
  return fx;
}
#undef ITMAX
#undef CGOLD
#undef ZEPS
#undef SHFT
