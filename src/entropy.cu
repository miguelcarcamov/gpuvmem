#include "entropy.cuh"

extern long M, N;
extern int image_count;
extern float *penalizators;
extern int nPenalizators;

Entropy::Entropy() {
  this->name = "Entropy";
  this->prior_value = 1.0f;
  this->eta = -1.0f;
};

Entropy::Entropy(float prior_value) {
  this->name = "Entropy";
  this->prior_value = prior_value;
  this->eta = -1.0f;
};

Entropy::Entropy(float prior_value, float eta) {
  this->name = "Entropy";
  this->prior_value = prior_value;
  this->eta = eta;
};

float Entropy::getPrior() { return this->prior_value; };

void Entropy::setPrior(float prior_value) { this->prior_value = prior_value; };

float Entropy::getEta() { return this->eta; };

void Entropy::setEta(float eta) { this->eta = eta; };

float Entropy::calcFi(float *p) {
  float result = 0.0f;
  this->set_fivalue(SEntropy(p, device_S, this->prior_value, this->eta,
                             penalization_factor, mod, order, imageIndex,
                             this->iteration));
  result = (penalization_factor) * (this->get_fivalue());
  return result;
};

void Entropy::calcGi(float *p, float *xi) {
  DEntropy(p, device_DS, this->prior_value, this->eta, penalization_factor, mod,
           order, imageIndex, this->iteration);
};

void Entropy::restartDGi() {
  checkCudaErrors(hipMemset(device_DS, 0, sizeof(float) * M * N));
};

void Entropy::addToDphi(float *device_dphi) {
  linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void Entropy::setSandDs(float *S, float *Ds) {
  hipFree(this->device_S);
  hipFree(this->device_DS);
  this->device_S = S;
  this->device_DS = Ds;
};

namespace {
Fi *CreateEntropy() { return new Entropy; }
const std::string name = "Entropy";
const bool RegisteredEntropy =
    registerCreationFunction<Fi, std::string>(name, CreateEntropy);
const bool RegisteredEntropyInt =
    registerCreationFunction<Fi, int>(0, CreateEntropy);
};  // namespace
