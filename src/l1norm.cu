#include "l1norm.cuh"

extern long M, N;
extern int image_count;
extern float* penalizators;
extern int nPenalizators;

L1norm::L1norm() {
  this->name = "L1 Norm";
  this->epsilon = 1E-12;
};

L1norm::L1norm(float epsilon) {
  this->name = "L1 Norm";
  this->epsilon = epsilon;
};

float L1norm::getEpsilon() {
  return this->epsilon;
};

void L1norm::setEpsilon(float epsilon) {
  this->epsilon = epsilon;
};

float L1norm::calcFi(float* p) {
  float result = 0.0f;
  this->set_fivalue(L1Norm(p, device_S, penalization_factor, this->epsilon, mod,
                           order, imageIndex, this->iteration));
  result = (penalization_factor) * (this->get_fivalue());
  return result;
}
void L1norm::calcGi(float* p, float* xi) {
  DL1Norm(p, device_DS, penalization_factor, this->epsilon, mod, order,
          imageIndex, this->iteration);
};

void L1norm::restartDGi() {
  checkCudaErrors(hipMemset(device_DS, 0, sizeof(float) * M * N));
};

void L1norm::addToDphi(float* device_dphi) {
  linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void L1norm::setSandDs(float* S, float* Ds) {
  hipFree(this->device_S);
  hipFree(this->device_DS);
  this->device_S = S;
  this->device_DS = Ds;
};

namespace {
Fi* CreateL1norm() {
  return new L1norm;
}
const std::string name = "L1-Norm";
const bool RegisteredL1norm =
    registerCreationFunction<Fi, std::string>(name, CreateL1norm);
};  // namespace
